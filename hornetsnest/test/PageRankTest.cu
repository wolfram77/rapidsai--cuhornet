#include "hip/hip_runtime.h"
/**
 * @brief PageRank test program
 * @file
 */
#include "Static/PageRank/PageRank.cuh"
#include <StandardAPI.hpp>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <iostream>
#include <fstream>
#include <string>

#include <cmath>
#include <vector>
#include <omp.h>

using namespace hornets_nest;




/**
 * Compute the L1-norm of the difference of two arrays in parallel.
 * @param x an array
 * @param y another array
 * @param N size of arrays
 * @param a initial value
 * @returns ||x-y||_1
 */
template <class TX, class TY, class TA=TX>
inline TA l1NormDeltaOmp(const TX *x, const TY *y, size_t N, TA a=TA()) {
  // ASSERT(x && y);
  #pragma omp parallel for schedule(auto) reduction(+:a)
  for (size_t i=0; i<N; ++i)
    a += TA(std::abs(x[i] - y[i]));
  return a;
}




int exec(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;
    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;


    graph::GraphStd<vid_t, eoff_t> graph;
    graph.read(argv[1], PRINT_INFO | SORT);
    // CommandLineParam cmd(graph, argc, argv);

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());
    HornetGraph hornet_graph(hornet_init);

    StaticPageRank page_rank(hornet_graph, 500, 1e-10, 0.85, false);

    Timer<DEVICE> TM;
    TM.start();

    page_rank.run();

    TM.stop();
    TM.print("PR---InputAsIS");

    // Retrieve pagerank values from device
    const pr_t *ranks = page_rank.get_page_rank_score_host();

    // Run reference PageRank
    StaticPageRank page_rank_ref(hornet_graph, 500, 0, 0.85, false);
    page_rank_ref.run();
    const pr_t *ranks_ref = page_rank_ref.get_page_rank_score_host();

    // Compare pagerank values with reference
    pr_t diff = l1NormDeltaOmp(ranks, ranks_ref, graph.nV());
    printf("Error: %.2e\n", diff);

	// graph::ParsingProp flag = PRINT_INFO | SORT;
	//         graph::GraphStd<vid_t, eoff_t> graphUnDir(UNDIRECTED);
  //   graphUnDir.read(argv[1],flag);

  //   HornetInit hornet_init_undir(graphUnDir.nV(), graphUnDir.nE(), graphUnDir.csr_out_offsets(),
  //                          graphUnDir.csr_out_edges());
  //   HornetGraph hornet_graph_undir(hornet_init_undir);

  //   StaticPageRank page_rank_undir(hornet_graph_undir, 500, 1e-10, 0.85, true);

  //   TM.start();

  //   page_rank_undir.run();

  //   TM.stop();
  //   TM.print("PR---Undirected---PULL");



    return 0;
}

int main(int argc, char* argv[]) {
  int ret = 0;
  {

    ret = exec(argc, argv);

  }

  return ret;
}
