/**
 * @brief PageRank test program
 * @file
 */
#include "Static/PageRank/PageRank.cuh"
#include <StandardAPI.hpp>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <iostream>
#include <fstream>
#include <string>

using namespace hornets_nest;

int exec(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;
    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;


    graph::GraphStd<vid_t, eoff_t> graph;
    graph.read(argv[1], PRINT_INFO | SORT);
    // CommandLineParam cmd(graph, argc, argv);

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());
    HornetGraph hornet_graph(hornet_init);

    StaticPageRank page_rank(hornet_graph, 500, 1e-10, 0.85, false);

    Timer<DEVICE> TM;
    TM.start();

    page_rank.run();

    TM.stop();
    TM.print("PR---InputAsIS");

    // Save pagerank values to file
    printf("Saving pagerank values to file %s.pr ...\n", argv[1]);
    std::ofstream ranksFile(argv[1] + std::string(".pr"));
    const pr_t *ranks = page_rank.get_page_rank_score_host();
    for (vid_t v = 0; v < graph.nV(); v++) {
        ranksFile << v << " " << ranks[v] << std::endl;
    }
    ranksFile.close();
    host::free(ranks);
    printf("Done\n");

	// graph::ParsingProp flag = PRINT_INFO | SORT;
	//         graph::GraphStd<vid_t, eoff_t> graphUnDir(UNDIRECTED);
  //   graphUnDir.read(argv[1],flag);

  //   HornetInit hornet_init_undir(graphUnDir.nV(), graphUnDir.nE(), graphUnDir.csr_out_offsets(),
  //                          graphUnDir.csr_out_edges());
  //   HornetGraph hornet_graph_undir(hornet_init_undir);

  //   StaticPageRank page_rank_undir(hornet_graph_undir, 500, 1e-10, 0.85, true);

  //   TM.start();

  //   page_rank_undir.run();

  //   TM.stop();
  //   TM.print("PR---Undirected---PULL");



    return 0;
}

int main(int argc, char* argv[]) {
  int ret = 0;
  {

    ret = exec(argc, argv);

  }

  return ret;
}
